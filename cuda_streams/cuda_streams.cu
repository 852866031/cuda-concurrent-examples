#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel(int *data, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long int startClock = clock64();
    // Spin until the desired duration has passed
    while (clock64() - startClock < 100000000) { //cycles to spin
        data[idx] += value;
    }
}

int main() {
    const int arraySize = 1024;
    const int blockSize = 256;
    const int numBlocks = arraySize / blockSize;

    int *d_data1, *d_data2;
    hipMalloc(&d_data1, arraySize * sizeof(int));
    hipMalloc(&d_data2, arraySize * sizeof(int));

    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch kernels in separate streams
    simpleKernel<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);
    simpleKernel<<<numBlocks, blockSize, 0, stream2>>>(d_data2, 20);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate and print elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time taken: " << milliseconds << " ms" << std::endl;

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(d_data1);
    hipFree(d_data2);

    return 0;
}
