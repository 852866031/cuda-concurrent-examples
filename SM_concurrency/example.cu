//Note that this example only works to show that you can run two kernels in one SM if your GPU only has one SM
//In a GPU with multiple SMs, you cannot explicity distribute blocks to SM 
// so use kernel 1 to occupy all SMs and almost all GPU resouces and let kernel 2 use only 1 thread does not work

#include <hip/hip_runtime.h>
#include <iostream>
__device__ volatile int s = 0;

__global__ void k1() {
    while (s == 0) {};  // Spin until s is non-zero
}

__global__ void k2() {
    s = 1;  // Set s to 1, allowing k1 to exit
}

int main() {
    // Create two separate streams
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    // Launch k1 in stream s1 and k2 in stream s2
    k1<<<1, 1, 0, s1>>>();
    k2<<<1, 1, 0, s2>>>();

    // Synchronize the device to wait for kernels to complete
    hipDeviceSynchronize();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    } else {
        std::cout << "Program terminated successfully." << std::endl;
    }

    return 0;
}
