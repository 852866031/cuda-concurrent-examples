
#include <hip/hip_runtime.h>
#include <iostream>
__device__ volatile int s = 0;

__global__ void k1() {
    long long int startClock = clock64(); // Record the starting clock cycle
    while (s == 0) {
        // Check if the maximum running time has been exceeded
        if (clock64() - startClock > 10000000000) {
            if(threadIdx.x == 0 && blockIdx.x == 0) printf("Kernel k1: Maximum running time exceeded!\n");
            return;
        }
    }
}

__global__ void k2() {
    // Set `s` to 1, allowing `k1` to exit
    s = 1;
}

int main() {
    // Get device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // Check the number of SMs and the maximum number of threads per block
    int numSMs = deviceProp.multiProcessorCount;
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    // Print device properties for debugging
    std::cout << "Number of SMs: " << numSMs << std::endl;
    std::cout << "Max threads per block: " << maxThreadsPerBlock << std::endl;

    // Create two separate streams
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    // Launch `k1` with enough blocks to occupy nearly all SMs but leave one block available
    int numBlocks = numSMs - 2; // Leave one block available for `k2`
    int threadsPerBlock = maxThreadsPerBlock/2; // Use the maximum number of threads per block

    // Launch `k1` in stream `s1`
    k1<<<numBlocks, threadsPerBlock, 0, s1>>>();

    // Launch `k2` in stream `s2`
    k2<<<1, 1, 0, s2>>>();

    // Synchronize the device to wait for kernels to complete
    hipDeviceSynchronize();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    } else {
        std::cout << "Program terminated successfully." << std::endl;
    }

    return 0;
}
