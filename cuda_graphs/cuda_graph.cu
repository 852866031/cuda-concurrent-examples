#include <hip/hip_runtime.h>
#include <iostream>

// Define four simple CUDA kernels
__global__ void A(int *data, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long int startClock = clock64();
    // Spin until the desired duration has passed
    while (clock64() - startClock < 1000000000) { // cycles to spin
        data[idx] += value;
    }
}

__global__ void B(int *data, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long int startClock = clock64();
    // Spin until the desired duration has passed
    while (clock64() - startClock < 1000000000) { // cycles to spin
        data[idx] += value;
    }
}

__global__ void C(int *data, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long int startClock = clock64();
    // Spin until the desired duration has passed
    while (clock64() - startClock < 1000000000) { // cycles to spin
        data[idx] += value;
    }
}

__global__ void D(int *data, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long int startClock = clock64();
    // Spin until the desired duration has passed
    while (clock64() - startClock < 1000000000) { // cycles to spin
        data[idx] += value;
    }
}

int main() {
    const int arraySize = 1024;
    const int blockSize = 256;
    const int numBlocks = arraySize / blockSize;
    int *d_data1;
    hipMalloc(&d_data1, arraySize * sizeof(int));

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipEvent_t e1, e2, startEvent, stopEvent, seqStartEvent, seqStopEvent;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&seqStartEvent);
    hipEventCreate(&seqStopEvent);


    // Measure time for CUDA Graph execution
    hipEventRecord(startEvent, stream1);
    hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);

    A<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);
    hipEventRecord(e1, stream1);
    B<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);

    hipStreamWaitEvent(stream2, e1);
    C<<<numBlocks, blockSize, 0, stream2>>>(d_data1, 10);
    hipEventRecord(e2, stream2);
    hipStreamWaitEvent(stream1, e2);
    D<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);

    hipGraph_t graph;
    hipStreamEndCapture(stream1, &graph);

    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);
    hipGraphLaunch(graphExec, stream1);
    hipEventRecord(stopEvent, stream1);
    hipStreamSynchronize(stream1);

    // Calculate and print elapsed time for CUDA Graph execution
    float graphMilliseconds = 0;
    hipEventElapsedTime(&graphMilliseconds, startEvent, stopEvent);
    std::cout << "Time taken for CUDA Graph execution: " << graphMilliseconds << " ms" << std::endl;

    // Measure time for sequential execution
    hipEventRecord(seqStartEvent, stream1);

    A<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);
    hipStreamSynchronize(stream1);
    B<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);
    hipStreamSynchronize(stream1);
    C<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);
    hipStreamSynchronize(stream1);
    D<<<numBlocks, blockSize, 0, stream1>>>(d_data1, 10);
    hipStreamSynchronize(stream1);

    hipEventRecord(seqStopEvent, stream1);
    hipStreamSynchronize(stream1);

    // Calculate and print elapsed time for sequential execution
    float seqMilliseconds = 0;
    hipEventElapsedTime(&seqMilliseconds, seqStartEvent, seqStopEvent);
    std::cout << "Time taken for sequential execution: " << seqMilliseconds << " ms" << std::endl;

    // Cleanup
    hipEventDestroy(e1);
    hipEventDestroy(e2);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipEventDestroy(seqStartEvent);
    hipEventDestroy(seqStopEvent);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(d_data1);

    std::cout << "CUDA Graph executed successfully." << std::endl;
    return 0;
}
