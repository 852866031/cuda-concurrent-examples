#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel1() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long int startClock = clock64();
    // Spin until the desired duration has passed
    while (clock64() - startClock < 100000000) { //cycles to spin
        data[idx] += value;
    }
}

int main() {
    kernel1<<<256, 256>>>();
    hipDeviceSynchronize();
    std::cout << "Kernel 1 completed." << std::endl;
    return 0;
}
