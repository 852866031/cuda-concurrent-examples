#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel2(int *data, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long int startClock = clock64();
    // Spin until the desired duration has passed
    while (clock64() - startClock < 10000000000) { //cycles to spin
        data[idx] += value;
    }
}

int main() {
    const int arraySize = 1024;
    const int blockSize = 256;
    const int numBlocks = arraySize / blockSize;

    int *d_data1;
    hipMalloc(&d_data1, arraySize * sizeof(int));
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);
    kernel2<<<numBlocks, blockSize>>>(d_data1, 10);
     // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time taken: " << milliseconds << " ms" << std::endl;
    std::cout << "Kernel 2 completed." << std::endl;
    return 0;
}